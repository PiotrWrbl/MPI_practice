#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

	__global__
	void checkPrime(int *isPrime, long inputArgument) {

	int my_index=blockIdx.x*blockDim.x+threadIdx.x;


		if (my_index<inputArgument) {

			if(inputArgument % my_index == 0) {
				isPrime[my_index]=1;
				}
			else{
			isPrime[my_index] = 0;
			}
		}
	}


int main(int argc, char **argv) {

  Args ins__args;
  parseArgs(&ins__args, &argc, argv);
  
  //program input argument
  long inputArgument = ins__args.arg; 

  struct timeval ins__tstart, ins__tstop;
  gettimeofday(&ins__tstart, NULL);
  
  // run your CUDA kernel(s) here
	
	long isPrime = 0 ;
	
	int threadsinblock=1024;
	int blocksingrid=10000;
	
	int size = threadsinblock*blocksingrid;
	
	
	int *hisPrime=(int*)malloc(size*sizeof(int));
	
	int *disPrime=NULL;
	
	hipMalloc((void**)&disPrime, size*sizeof(int));
	
	
	checkPrime <<<blocksingrid,threadsinblock>>> (disPrime, inputArgument);
	
	
	hipMemcpy(hisPrime, disPrime, size*sizeof(int),hipMemcpyDeviceToHost);
	 
	
	for(int i=0 ;i<size; i++){
	isPrime = isPrime + hisPrime[i]; 
	}
	
	free(hisPrime);
	hipFree(disPrime);
	
    // synchronize/finalize your CUDA computations

  gettimeofday(&ins__tstop, NULL);
  ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);

	if (isPrime<=2){
		printf("%ld is a prime number. \n", inputArgument);
		}
		
  else {
		printf("%ld is not a prime number. /n",inputArgument);
		}
		
}
